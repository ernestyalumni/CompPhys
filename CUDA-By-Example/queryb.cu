/*
 * queryb.cu
 */
/* cf. Jason Sanders, Edward Kandrot. CUDA by Example: An Introduction to General-Purpose GPU Programming */
/* 3.3 Querying Devices 
** pp. 32 */
#include <stdio.h>
#include "common/errors.h"

int main(void) {
  hipDeviceProp_t prop;

  int count;
  HANDLE_ERROR(
	       hipGetDeviceCount( &count )
	       );

  for (int i = 0 ; i < count ; i++ ) {
    HANDLE_ERROR(
		 hipGetDeviceProperties( &prop, i )
		 );
    printf("    --- General Information for device %d ---\n", i);
    printf( "Name: %s\n", prop.name );
    printf( "Compute capability: %d.%d\n", prop.major, prop.minor );
    printf( "Clock rate:  %d\n", prop.clockRate);
    printf( "Device copy overlap:  ");
    if (prop.deviceOverlap)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n" );
    printf( "Kernel execution timeout :  ");
    if (prop.kernelExecTimeoutEnabled)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n" );

    printf("   --- Memory Information for device %d ---\n", i );
    printf("Total global mem:      %ld\n", prop.totalGlobalMem );
    printf("Total constant Mem:    %ld\n", prop.totalConstMem  );
    printf("Max mem pitch:         %ld\n", prop.memPitch );
    printf("Texture Alignment:     %ld\n", prop.textureAlignment);
    printf("   --- MP Information for device %d ---\n", i);
    printf("Multiprocessor count:  %d\n",
	   prop.multiProcessorCount);
    printf("Shared mem per mp:     %ld\n", prop.sharedMemPerBlock);
    printf("Registers per mp:      %d\n",  prop.regsPerBlock);
    printf("Threads in warp:       %d\n",  prop.warpSize);
    printf("Max threads per block: %d\n",
	   prop.maxThreadsPerBlock );
    printf("Max thread dimensions: (%d, %d, %d) \n",
	   prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	   prop.maxThreadsDim[2]);
    printf("Max grid dimensions:   (%d, %d, %d) \n",
	   prop.maxGridSize[0], prop.maxGridSize[1],
	   prop.maxGridSize[2] );
    printf("\n");
    
    printf("   --- Other Information for device %d ---\n", i);
    printf("Max. 3D textures dimensions: (%d, %d, %d) \n",
	   prop.maxTexture3D[0], prop.maxTexture3D[1], prop.maxTexture3D[2] );
  }
}
